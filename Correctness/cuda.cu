
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <math.h>
/*Matrix size N*/
#define N  20

#define CHECK_ERR(x)                                    \
if (x != hipSuccess) {                               \
fprintf(stderr,"%s in %s at line %d\n",             \
cudaGetErrorString(err),__FILE__,__LINE__);     \
exit(-1);                                               \
}                                                     \
/*host variables for matrices*/
float h_A[N][N];
float h_B[N][N];

__global__ void func ( float *temp_d, float *mean_d, float *SD_d, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n )
        /*d_A[j][i] = temp_d[i];
        /*computations*/
        if(SD_d[i] == 0.0){
            temp_d[i] = 0.0;
        }else{
            temp_d[i] = (temp_d[i] - mean_d[i]) / SD_d[i];
    }
}
/*initializing matrix with random values*/
void initialize_inputs() {
    int row, col;
    srand((unsigned)time(NULL));
    for (row = 0; row < N; row++) {
        for (col = 0; col < N; col++) {
            h_A[row][col] = row+1;
            h_B[row][col] = 0.0;
        }
    }
}

int main() {

    hipError_t err;
    int i,j;

    /*host variables*/
    float transpose_A[N][N];
    float temp2[N];
    float temp_h[N];

    /* Mean and Standard Deviation variables */
    float mean[N];
    float SD[N];
    int row, col;
    float mu, sigma;

    /*device variables*/
    float d_A[N][N];
    float out_A[N][N];
    float *temp_d;
    float *mean_d;
    float *SD_d;

    /*timing variables*/
    struct timeval start, stop;  
    struct timezone tzdummy;
    unsigned long long runtime;
    unsigned long long total_time = 0;

    /*Program begins*/
    initialize_inputs();

    /*compute sd anad mean for every column*/
    for (col=0; col < N; col++) {
        mu = 0.0;
        for (row=0; row < N; row++)
            mu += h_A[row][col];
        mu /= (float) N;
        sigma = 0.0;
        for (row=0; row < N; row++)
            sigma += powf(h_A[row][col] - mu, 2.0);
        sigma /= (float) N;
        sigma = sqrt(sigma);

        mean[col] = mu;
        SD[col] = sigma;
    }
    /*testing */
    printf("\nAll the means :\n");
    for(i=0;i<N;i++){
        printf("%.2f ",mean[i]);
    }
    printf("\nAll the SDs :\n");
    for(i=0;i<N;i++){
        printf("%.2f ",SD[i]);
    }

    /*transpose matrix so that cols are rows*/
    for (i = 0; i < N; i++){
      for(j = 0 ; j < N ; j++){
         transpose_A[j][i] = h_A[i][j];
      }
    }
    /**testing*/
    printf("\nThe transpose is\n");
    for (i = 0; i < N; i++){
      for(j = 0 ; j < N ; j++){
         printf("%.2f ",transpose_A[i][j]);
      }
    }
    /*********************Allocate memory on device**********************/
    /*allocate temp_d on device*/
    err = hipMalloc((void **) &temp_d, sizeof(float)*N);
    CHECK_ERR(err);

    /*allocate memory for every column on device*/
    for(i=0;i<N;i++){
        err = hipMalloc((void **) &d_A[i], sizeof(float)*N);
        CHECK_ERR(err);
    }

    /*allocate memory for means array on device*/
    err = hipMalloc((void **) &mean_d, sizeof(float)*N);
    CHECK_ERR(err);
    /*allocate memory for SDs array on device*/
    err = hipMalloc((void **) &SD_d, sizeof(float)*N);
    CHECK_ERR(err);

    /**********************Send data to GPU********************************/
    /*sending means array */
    err = hipMemcpy(mean_d, mean, sizeof(float)*N, hipMemcpyHostToDevice);
    CHECK_ERR(err);
    /*sending SDs array */
    err = hipMemcpy(SD_d, SD, sizeof(float)*N, hipMemcpyHostToDevice);
    CHECK_ERR(err);
    /*sending columns of the matrix*/

    for(i=0;i<N;i++){
        for(j=0;j<N;j++){
            temp_h[j] = transpose_A[i][j];
        }
        err = hipMemcpy(temp_d, temp_h, sizeof(float)*N, hipMemcpyHostToDevice);
        CHECK_ERR(err);
        j=0;
        /********** Start Clock ***********/
        hipDeviceSynchronize();
        gettimeofday(&start, &tzdummy);

        func<<<ceil(N/256.0), 256>>>(temp_d,mean_d,SD_d,N);
        /************stop clock************/
        hipDeviceSynchronize();
        gettimeofday(&stop, &tzdummy);
        runtime = (unsigned long long)(stop.tv_sec - start.tv_sec) * 1000000 + (stop.tv_usec - start.tv_usec);

        total_time = total_time + runtime;
        err = hipMemcpy(temp2, temp_d, sizeof(float)*N, hipMemcpyDeviceToHost);

        for(j=0;j<N;j++){
            out_A[i][j] = temp2[j];
        }
    }
    /*************************Print output**********************************/
    printf("\nAfter Normalization :\n");
    for(i=0;i<N;i++){
        for(j=0;j<N;j++){
            /*printf("%f  ", out_A[i][j] );*/
            printf("%5.2f%s", out_A[j][i], (j < N-1) ? ", " : ";\n\t");
        }
    }

    /* Display timing results */
    printf("Runtime = %g ms.\n", (float)total_time/(float)1000);
    printf("\nStopped clock.");
    printf("\n---------------------------------------------\n");

}
